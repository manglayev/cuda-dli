
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);

  hipError_t err_1;
  err_1 = hipMallocManaged(&a, size);                    // Assume the existence of `a` and `N`.

  if (err_1 != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
      printf("Error: %s\n", hipGetErrorString(err_1)); // `cudaGetErrorString` is provided by CUDA.
  }

  init(a, N);

  size_t threads_per_block = 2048;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipError_t err_2;
  err_2 = hipGetLastError(); // `cudaGetLastError` will return the error from above.
  if (err_2 != hipSuccess)
  {
      printf("Error: %s\n", hipGetErrorString(err_2));
  }

  hipDeviceSynchronize();
  hipError_t err_3;
  err_3 = hipGetLastError(); // `cudaGetLastError` will return the error from above.
  if (err_3 != hipSuccess)
  {
      printf("Error: %s\n", hipGetErrorString(err_3));
  }

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
